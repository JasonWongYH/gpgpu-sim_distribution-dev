#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_fp16.h>
#include <assert.h>
#include "fp16_conversion.h"   // host function for half conversion

#define TWO_PI             6.28318530717959f
#define TWO_PI_H	__float2half(6.28318530717959f)
//#define TWO_PI             6.28

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__global__
void myTest(int n, float a, const float *x, half *y)
{
	int gid = threadIdx.x + blockDim.x * blockIdx.x;

	half a_half = __float2half(a);
	half2 a2 = __halves2half2(a_half, a_half);

	half one_half = __float2half(1.f);
	half2 one2 = __halves2half2( __float2half(1.f), __float2half(2.f) );

	// half2 add
	// result.low = 1
	// result.high = 2
	half2 result = __hadd2(a2, one2);

	if(gid<n) {
		y[gid] = __low2half(result);  // 1 + 1
		//y[gid] = __high2half(result);  // 1 + 2
	}
}

int main(int argc, char** argv) {

  int devid = atoi(argv[1]);
  hipSetDevice(devid);

  hipDeviceProp_t prop;                                                    
  hipGetDeviceProperties(&prop, devid);                                 
  printf("device %d : %s\n", devid, prop.name);

  const int n = 32;

  const float a = 1.f; 
  printf("a = %f\n", a);

  float *x;
  checkCuda(hipMallocManaged(&x, n * sizeof(float)));

  half *y;
  checkCuda(hipMallocManaged(&y, n * sizeof(half)));
  
  for (int i = 0; i < n; i++) {
    x[i] = 1.0f;
    y[i] = approx_float_to_half(2.f);
  }


  const int blockSize = 32;
  const int nBlocks = (n + blockSize - 1) / blockSize;

  myTest<<<nBlocks, blockSize>>>(n, a, x, y);

  // must wait for kernel to finish before CPU accesses
  checkCuda(hipDeviceSynchronize());
  
  for (int i = 0; i < n; i++)
  	printf("%f\n", half_to_float(y[i]));


  return 0;
}
