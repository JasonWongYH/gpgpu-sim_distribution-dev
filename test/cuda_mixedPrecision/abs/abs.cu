#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_fp16.h>
#include <assert.h>
#include "fp16_conversion.h"   // host function for half conversion

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__device__ inline half h_fabs(half v) {
    return (__hgeu(v,__float2half(0.0f))?v:__hneg(v));
}

__global__
void myTest(float *x, half *y)
{
    y[0] = h_fabs(__float2half(x[0]));
}

__global__
void myTest_v1(float *x, float *y)
{
    y[0] = __half2float(h_fabs(__float2half(x[0])));
}



void mytest(float v)
{
  float *x;
  checkCuda(hipMallocManaged(&x, sizeof(float)));

  half *y;
  checkCuda(hipMallocManaged(&y, sizeof(half)));

  x[0] = v;
  printf("input\t : %f\n", x[0]);
  
  myTest<<<1, 1>>>(x, y);

  checkCuda(hipDeviceSynchronize());
  
  // check results
  printf("output\t : %f\n", half_to_float(y[0]));

  hipFree(x);
  hipFree(y);
}


void mytest_v1(float v)
{
  float *x;
  checkCuda(hipMallocManaged(&x, sizeof(float)));

  float *y;
  checkCuda(hipMallocManaged(&y, sizeof(float)));

  x[0] = v;
  printf("input\t : %f\n", x[0]);

  half tmpV = approx_float_to_half(x[0]);
  printf("input (approx half)\t : %f\n", tmpV );
  printf("input (approx half, back to float)\t : %f\n", half_to_float(tmpV) );
  
  myTest_v1<<<1, 1>>>(x, y);

  checkCuda(hipDeviceSynchronize());
  
  // check results
  printf("output(native __half2float)\t : %f\n", y[0]);

  hipFree(x);
  hipFree(y);
}

__global__ void Kern_half2_abs(float *x)
{
	half2 v2 = __floats2half2_rn(1.1f, -2.1f);

	printf("value in half2 : %f %f\n", __half2float(__low2half(v2)),
			__half2float(__high2half(v2)));

	// abs()
	*((int*)(&v2)) &= 0x7FFF7FFF;

	printf("value in half2 (abs) : %f %f\n", __half2float(__low2half(v2)),
			__half2float(__high2half(v2)));

}


void half2_abs(void)
{
  float *x;
  checkCuda(hipMallocManaged(&x, sizeof(float)));

  Kern_half2_abs<<<1, 1>>>(x);


  checkCuda(hipDeviceSynchronize());
  hipFree(x);
}

int main(int argc, char** argv) {

  int devid =0;

  if (argc != 2) {
      fprintf(stderr, "Specify device to use only. (./program devid)\n");
      exit(1);
  }else {
  	devid = atoi(argv[1]);
  }

  hipSetDevice(devid);

  hipDeviceProp_t prop;                                                    
  hipGetDeviceProperties(&prop, devid);                                 
  printf("device %d : %s\n", devid, prop.name);


  /*
  printf("\ntest 1\n");
  float v_pos = 1.111f;
  float v_neg = -1.111f;

  mytest(v_pos);
  mytest(v_neg);
  mytest_v1(v_pos);
  mytest_v1(v_neg);

  printf("\ntest 2\n");
  v_pos = 1.11f;
  v_neg = -1.11f;

  mytest(v_pos);
  mytest(v_neg);
  mytest_v1(v_pos);
  mytest_v1(v_neg);

  printf("\ntest 3\n");
  v_pos = 1.3333f;
  v_neg = -1.3333f;

  mytest(v_pos);
  mytest(v_neg);
  mytest_v1(v_pos);
  mytest_v1(v_neg);
  */


  half2_abs();

  return 0;
}
