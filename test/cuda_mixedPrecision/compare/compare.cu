#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_fp16.h>
#include <assert.h>
#include "fp16_conversion.h"   // host function for half conversion

#define TWO_PI             6.28318530717959f
#define TWO_PI_H	__float2half(6.28318530717959f)
//#define TWO_PI             6.28

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}

__global__
void myTest(int n, float a, const float *x, half *y)
{
/*
	half a_half = __float2half(a);
	y[0] = __hadd(a_half, __hge(a_half, bool2half(__float2half(0.f)))); 
*/



	half2 zero2 = __floats2half2_rn(0.f, 0.f);
	half2 minusone = __floats2half2_rn(-1.f, -1.f);
	half2 v2    = __floats2half2_rn(2.1f, -3.1f);
	printf("v2:  %f %f\n", __half2float(__low2half(v2)),  __half2float(__high2half(v2)));
	// compare 
	half2 cmp2 = __hgt2(v2, zero2);
	printf("(compare >0.f) %f %f\n", __half2float(__low2half(cmp2)),  __half2float(__high2half(cmp2)));
	// abs 
	half2 r2 = __hmul2(__hadd2(__hadd2(cmp2, cmp2), minusone), v2);
	printf("(abs)%f %f\n", __half2float(__low2half(r2)),  __half2float(__high2half(r2)));
}

int main(int argc, char** argv) {

  int devid = atoi(argv[1]);
  hipSetDevice(devid);

  hipDeviceProp_t prop;                                                    
  hipGetDeviceProperties(&prop, devid);                                 
  printf("device %d : %s\n", devid, prop.name);

  const int n = 1;

  const float a = 1.f; 
  //printf("a = %f\n", a);

  float *x;
  checkCuda(hipMallocManaged(&x, n * sizeof(float)));

  half *y;
  checkCuda(hipMallocManaged(&y, n * sizeof(half)));
  
  for (int i = 0; i < n; i++) {
    x[i] = 1.0f;
    y[i] = approx_float_to_half(2.f);
  }


  myTest<<<1, 1>>>(n, a, x, y);

  checkCuda(hipDeviceSynchronize());
  
/*
  for (int i = 0; i < n; i++)
  	printf("%f\n", half_to_float(y[i]));
*/


  return 0;
}
